
#include <hip/hip_runtime.h>
#include <iostream>
/*
#include <chrono>

// helper functions for cleaner time measuring code
std::chrono::time_point<std::chrono::high_resolution_clock> now() {
    return std::chrono::high_resolution_clock::now();
}

template <typename T>
double milliseconds(T t) {
    return (double) std::chrono::duration_cast<std::chrono::nanoseconds>(t).count() / 1000000;
}
*/
// gpu kernel function
__global__
void addKernel(double* x, double* y, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

extern "C"
double* addVectorsGPU(double* a, double* b, int n) {
  //  auto t1 = now();

    double* x;
    double* y;
    double* z;
    hipMalloc(&x, n * sizeof(double));
    hipMalloc(&y, n * sizeof(double));
    hipMemcpy(x, a, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y, b, n * sizeof(double), hipMemcpyHostToDevice);

    //auto t2 = now();

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = deviceProp.maxThreadsPerBlock;
    int numBlocks = (n - 1) / blockSize + 1;

    addKernel<<<numBlocks, blockSize>>>(x, y, n);
    hipDeviceSynchronize();

    //auto t3 = now();

    z = (double*) malloc(n * sizeof(double));
    hipMemcpy(z, y, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(x);
    hipFree(y);
/*
    auto t4 = now();

    std::cout << "GPU time breakdown--------\n";
    std::cout << "loading into device memory: " << milliseconds(t2 - t1) << " milliseconds\n";
    std::cout << "actual addition:            " << milliseconds(t3 - t2) << " milliseconds\n";
    std::cout << "loading into host memory:   " << milliseconds(t4 - t3) << " milliseconds\n";
*/
    return z;
}
